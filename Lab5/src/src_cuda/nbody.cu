#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_fp16.h>
#include "timer.h"

#define SOFTENING 1e-9f  /* Will guard against denormals */
#define THREADS_PER_BLOCK 1024

typedef struct { __half *x, *y, *z, *vx, *vy, *vz;} Body;

/****************************** Helper Functions ******************************/
void checkCudaError(const char *step) {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error in %s: %s\n", step, hipGetErrorString(err));
		hipDeviceReset();
		exit(1);
	}
}

void randomizeBodies(Body *bodies, int n) {
  	for (int i = 0; i < n; i++) {
    	bodies->x[i]  = __float2half(2.0f * (rand() / (float)RAND_MAX) - 1.0f);
    	bodies->y[i]  = __float2half(2.0f * (rand() / (float)RAND_MAX) - 1.0f);
    	bodies->z[i]  = __float2half(2.0f * (rand() / (float)RAND_MAX) - 1.0f);
    	bodies->vx[i] = __float2half(2.0f * (rand() / (float)RAND_MAX) - 1.0f);
    	bodies->vy[i] = __float2half(2.0f * (rand() / (float)RAND_MAX) - 1.0f);
    	bodies->vz[i] = __float2half(2.0f * (rand() / (float)RAND_MAX) - 1.0f);
  	}
}

/***************** KERNEL CODE *****************/
__global__ void bodyForce(Body p, __half dt, int tiles, int n) {
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int tile;
	
	__half dx, dy, dz;
	__half distSqr, invDist, invDist3;
	__half Fx = __float2half(0.0f);
	__half Fy = __float2half(0.0f);
	__half Fz = __float2half(0.0f);

	__shared__ __half body_coordinates_x[THREADS_PER_BLOCK];
	__shared__ __half body_coordinates_y[THREADS_PER_BLOCK];
	__shared__ __half body_coordinates_z[THREADS_PER_BLOCK];
	__half curr_x = p.x[tid];
	__half curr_y = p.y[tid];
	__half curr_z = p.z[tid];

	if (tid == 0) {
		printf("Coordinates: %f\n", __half2float(p.x[tid]));
		printf("Coordinates: %f\n", __half2float(p.y[tid]));
		printf("Coordinates: %f\n", __half2float(p.z[tid]));
	}
	
	int iter = 1;
	for (tile = 0; tile < tiles-1; tile++) {
		body_coordinates_x[threadIdx.x] = p.x[threadIdx.x + tile*blockDim.x];
		body_coordinates_y[threadIdx.x] = p.y[threadIdx.x + tile*blockDim.x];
		body_coordinates_z[threadIdx.x] = p.z[threadIdx.x + tile*blockDim.x];

		__syncthreads();
		// #pragma unroll 16
		for (int i = 0; i < THREADS_PER_BLOCK; i++) {
			dx = __hsub(body_coordinates_x[i], curr_x);
			dy = __hsub(body_coordinates_y[i], curr_y);
			dz = __hsub(body_coordinates_z[i], curr_z);
			distSqr = __hfma(dx, dx, __hfma(dy, dy, __hfma(dz, dz, __float2half(SOFTENING))));
			invDist = hrcp(hsqrt(distSqr));
			invDist3 = __hmul(invDist, __hmul(invDist, invDist));

			if (tid == 0 && iter == 1) {
				printf("dx: %f\n", __half2float(dx));
				printf("dy: %f\n", __half2float(dy));
				printf("dz: %f\n", __half2float(dz));
				printf("distSqr:     %f\n", __half2float(distSqr));
				printf("Invdist:     %f\n", __half2float(invDist));
				printf("Invdist3:    %f\n", __half2float(invDist3));

				iter = 2;
			}

			Fx = __hadd(Fx, __hmul(dx, invDist3)); 
			Fy = __hadd(Fy, __hmul(dy, invDist3)); 
			Fz = __hadd(Fz, __hmul(dz, invDist3));
		}
		__syncthreads();
	}

	// Bring last tile into shared memory;
	body_coordinates_x[threadIdx.x] = p.x[threadIdx.x + (tiles-1)*blockDim.x];
	body_coordinates_y[threadIdx.x] = p.y[threadIdx.x + (tiles-1)*blockDim.x];
	body_coordinates_z[threadIdx.x] = p.z[threadIdx.x + (tiles-1)*blockDim.x];
	__syncthreads();

	int last_bodies = (n%THREADS_PER_BLOCK == 0) ? THREADS_PER_BLOCK : n%THREADS_PER_BLOCK;

	// #pragma unroll 16
	for (int i = 0; i < last_bodies; i++) {
		dx = __hsub(body_coordinates_x[i], curr_x);
		dy = __hsub(body_coordinates_y[i], curr_y);
		dz = __hsub(body_coordinates_z[i], curr_z);
		distSqr = __hfma(dx, dx, __hfma(dy, dy, __hfma(dz, dz, __float2half(SOFTENING))));
		invDist = hrcp(hsqrt(distSqr));
		invDist3 = __hmul(invDist, __hmul(invDist, invDist));

		Fx = __hadd(Fx, __hmul(dx, invDist3)); 
		Fy = __hadd(Fy, __hmul(dy, invDist3)); 
		Fz = __hadd(Fz, __hmul(dz, invDist3));
	}

    p.vx[tid] = __hadd(p.vx[tid], __hmul(dt, Fx));
	p.vy[tid] = __hadd(p.vy[tid], __hmul(dt, Fy));
	p.vz[tid] = __hadd(p.vz[tid], __hmul(dt, Fz));
}

__global__ void calculatePositions(Body p, __half dt, int n) {
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	if (tid >= n) {
		return;
	}

	p.x[tid] = __hadd(p.x[tid], __hmul(p.vx[tid], dt));
	p.y[tid] = __hadd(p.y[tid], __hmul(p.vy[tid], dt));
	p.z[tid] = __hadd(p.z[tid], __hmul(p.vz[tid], dt));
}

int main(const int argc, const char** argv) {

  	int nBodies = 30000;
  	if (argc > 1) nBodies = atoi(argv[1]);

  	const __half dt = __float2half(0.01f); // time step
  	const int nIters = 10;  // simulation iterations

	float totalTime = 0.0f, elapsed_time = 0.0f;
	Body bodies, d_bodies;

	hipEvent_t iter_start, iter_end;

	hipEventCreate(&iter_start);
	hipEventCreate(&iter_end);

	/****************************** Host memory allocation ******************************/
	int bytes = sizeof(__half)*nBodies;
	bodies.x =  (__half *)malloc(bytes);
	bodies.y =  (__half *)malloc(bytes);
	bodies.z =  (__half *)malloc(bytes);
	bodies.vx = (__half *)malloc(bytes);
	bodies.vy = (__half *)malloc(bytes);
	bodies.vz = (__half *)malloc(bytes);

  	randomizeBodies(&bodies, nBodies); // Init pos / vel data

	// Set geometry
	dim3 block(THREADS_PER_BLOCK, 1, 1);
	dim3 grid((int)(ceil((float)nBodies/THREADS_PER_BLOCK)), 1, 1);
	int tiles = (int)(ceil((float)nBodies/THREADS_PER_BLOCK));

	/****************************** Data transfers ******************************/
	hipMalloc((void **) &d_bodies.x, bytes);
	hipMalloc((void **) &d_bodies.y, bytes);
	hipMalloc((void **) &d_bodies.z, bytes);
	hipMalloc((void **) &d_bodies.vx, bytes);
	hipMalloc((void **) &d_bodies.vy, bytes);
	hipMalloc((void **) &d_bodies.vz, bytes);

	/****************************** Real Computation ******************************/
  	for (int iter = 1; iter <= nIters; iter++) {
		hipEventRecord(iter_start, 0);

		// In the first iteration both initial coordinates and velocity needs to be copied to device
		if (iter == 1) {
			hipMemcpy(d_bodies.x,  bodies.x,  bytes, hipMemcpyHostToDevice);
			hipMemcpy(d_bodies.y,  bodies.y,  bytes, hipMemcpyHostToDevice);
			hipMemcpy(d_bodies.z,  bodies.z,  bytes, hipMemcpyHostToDevice);

			hipMemcpy(d_bodies.vx, bodies.vx, bytes, hipMemcpyHostToDevice);
			hipMemcpy(d_bodies.vy, bodies.vy, bytes, hipMemcpyHostToDevice);
			hipMemcpy(d_bodies.vz, bodies.vz, bytes, hipMemcpyHostToDevice);
		}

		bodyForce<<<grid, block>>>(d_bodies, dt, tiles, nBodies);
		checkCudaError("bodyForce");
        hipDeviceSynchronize();

		// hipMemcpy(bodies.vx, d_bodies.vx, bytes, hipMemcpyDeviceToHost);
		// hipMemcpy(bodies.vy, d_bodies.vy, bytes, hipMemcpyDeviceToHost);
		// hipMemcpy(bodies.vz, d_bodies.vz, bytes, hipMemcpyDeviceToHost);

		// printf("Iteration %d\n", iter);
		// for (int i = 0; i < 10; i++) {
		// 	printf("%f\n", __half2float(bodies.vx[i]));
		// 	printf("%f\n", __half2float(bodies.vy[i]));
		// 	printf("%f\n", __half2float(bodies.vz[i]));
		// }


		calculatePositions<<<grid, block>>>(d_bodies, dt, nBodies);
		checkCudaError("calculatePositions");
		hipDeviceSynchronize();

		// hipMemcpy(bodies.x, d_bodies.x, bytes, hipMemcpyDeviceToHost);
		// hipMemcpy(bodies.y, d_bodies.y, bytes, hipMemcpyDeviceToHost);
		// hipMemcpy(bodies.z, d_bodies.z, bytes, hipMemcpyDeviceToHost);

		// printf("Iteration %d\n", iter);
		// for (int i = 0; i < 10; i++) {
		// 	printf("%f\n", __half2float(bodies.x[i]));
		// 	printf("%f\n", __half2float(bodies.y[i]));
		// 	printf("%f\n", __half2float(bodies.z[i]));
		// }

		// Send final coordinates back to host
		if (iter == nIters) {
			hipMemcpy(bodies.x, d_bodies.x, bytes, hipMemcpyDeviceToHost);
			hipMemcpy(bodies.y, d_bodies.y, bytes, hipMemcpyDeviceToHost);
			hipMemcpy(bodies.z, d_bodies.z, bytes, hipMemcpyDeviceToHost);

		}

		/****************************** Save Final Coordinates ******************************/
		#ifdef SAVE_FINAL_COORDINATES
		if (iter == 2) {
			hipMemcpy(bodies.x, d_bodies.x, bytes, hipMemcpyDeviceToHost);
			hipMemcpy(bodies.y, d_bodies.y, bytes, hipMemcpyDeviceToHost);
			hipMemcpy(bodies.z, d_bodies.z, bytes, hipMemcpyDeviceToHost);
			char filename[256];
		
			sprintf(filename, "cuda_coordinates_%d.txt", nBodies);
		
			printf("Writing final coordinates to %s\n", filename);
			FILE *fd = fopen(filename, "w");
		
			if (!fd) {
				perror("Failed opening file");
				return -1;
			}
		
			for (int i = 0; i < nBodies; i++) {
				fprintf(fd, "%f\n", __half2float(bodies.x[i]));
				fprintf(fd, "%f\n", __half2float(bodies.y[i]));
				fprintf(fd, "%f\n", __half2float(bodies.z[i]));
			}
		
			fclose(fd);
		
			printf("Data written successfully\n");
		}
		#endif

        hipEventRecord(iter_end, 0);
		hipEventSynchronize(iter_end);

		hipEventElapsedTime(&elapsed_time, iter_start, iter_end);
    	if (iter > 1) { // First iter is warm up
      		totalTime += elapsed_time/1000.0f;
    	}
    	printf("Iteration %d: %.3f seconds\n", iter, elapsed_time/1000.0f);
  	}

  	float avgTime = totalTime / (float)(nIters-1);

  	printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, 1e-9 * nBodies * nBodies / avgTime);
	printf("Total time: %.3f\n", totalTime);

	/****************************** Cleanup ******************************/
	// Device
	hipFree(d_bodies.x);
	hipFree(d_bodies.y);
	hipFree(d_bodies.z);
	hipFree(d_bodies.vx);
	hipFree(d_bodies.vy);
	hipFree(d_bodies.vz);	
	
	// Host
	free(bodies.x);
	free(bodies.y);
	free(bodies.z);
	free(bodies.vx);
	free(bodies.vy);
	free(bodies.vz);	

	hipDeviceReset();
}