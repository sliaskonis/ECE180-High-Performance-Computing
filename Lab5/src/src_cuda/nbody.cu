#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"

#define SOFTENING 1e-9f  /* Will guard against denormals */
#define THREADS_PER_BLOCK 1024

typedef struct { float x, y, z, vx, vy, vz;} Body;

/****************************** Helper Functions ******************************/
bool checkCudaError(const char *step) {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error in %s: %s\n", step, hipGetErrorString(err));
		return true;
	}
	return false;
}

void randomizeBodies(float *data, int n) {
  	for (int i = 0; i < n; i++) {
    	data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  	}
}

/***************** KERNEL CODE *****************/
__global__ void bodyForce(Body *p, float dt, int tiles, int n) {
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int tile;

	float dx, dy, dz;
	float distSqr, invDist, invDist3;
	float Fx = 0.0f;
	float Fy = 0.0f;
	float Fz = 0.0f;

	__shared__ Body private_bodies[THREADS_PER_BLOCK];
	Body curr_body = p[tid];

	for (tile = 0; tile < tiles; tile++) {
		private_bodies[threadIdx.x] = p[threadIdx.x + tile * blockDim.x];
		__syncthreads();
		for (int i = 0; i < THREADS_PER_BLOCK; i++) {
			dx = private_bodies[i].x - curr_body.x;
			dy = private_bodies[i].y - curr_body.y;
			dz = private_bodies[i].z - curr_body.z;
			distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
			invDist = 1.0f / sqrtf(distSqr);
			invDist3 = invDist * invDist * invDist;

			Fx += dx * invDist3;
			Fy += dy * invDist3; 
			Fz += dz * invDist3;
		}
		__syncthreads();
	}

	// // Load last tile into shared memory
	// private_bodies[threadIdx.x] = p[threadIdx.x + (tiles-1) * blockDim.x];
	// __syncthreads();

	// int last_bodies = (n%THREADS_PER_BLOCK == 0) ? THREADS_PER_BLOCK : (THREADS_PER_BLOCK - n%THREADS_PER_BLOCK);

	// for (int j = 0; j < last_bodies; j++) {
	// 	dx = private_bodies[j].x - curr_body.x;
	// 	dy = private_bodies[j].y - curr_body.y;
	// 	dz = private_bodies[j].z - curr_body.z;
	// 	distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
	// 	invDist = 1.0f / sqrtf(distSqr);
	// 	invDist3 = invDist * invDist * invDist;

	// 	Fx += dx * invDist3;
	// 	Fy += dy * invDist3; 
	// 	Fz += dz * invDist3;
	// }

    curr_body.vx += dt*Fx;
	curr_body.vy += dt*Fy;
	curr_body.vz += dt*Fz;

	// Update global memory
	p[tid] = curr_body;
}

int main(const int argc, const char** argv) {

  	int nBodies = 30000;
  	if (argc > 1) nBodies = atoi(argv[1]);

  	const float dt = 0.01f; // time step
  	const int nIters = 10;  // simulation iterations

  	int bytes = nBodies*sizeof(Body);
	float totalTime = 0.0f, elapsed_time = 0.0f;
	float *buf = (float*)malloc(bytes);
	float *d_buf;
	Body *d_p, *p = (Body*)buf;
	hipEvent_t iter_start, iter_end;

	hipEventCreate(&iter_start);
	hipEventCreate(&iter_end);

  	randomizeBodies(buf, 6*nBodies); // Init pos / vel data

	// Set geometry
	dim3 block(THREADS_PER_BLOCK, 1, 1);
	dim3 grid((int)(ceil(nBodies/THREADS_PER_BLOCK)), 1, 1);
	int tiles = (int)(ceil(nBodies/THREADS_PER_BLOCK));

	/****************************** Data transfers ******************************/
	hipMalloc((void **) &d_buf, bytes);

	d_p = (Body*)d_buf;

	/****************************** Real Computation ******************************/
  	for (int iter = 1; iter <= nIters; iter++) {
		hipEventRecord(iter_start, 0);

		// Tranfer new coordinates back to device for next computations
		hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
		
		bodyForce<<<grid, block>>>(d_p, dt, tiles, nBodies);
		checkCudaError("bodyForce");
        hipDeviceSynchronize();

        // Transfer data back to host in order to compute new coordinates
	    hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);

		// Calculate new coordinates
        for (int i = 0 ; i < nBodies; i++) {
            p[i].x += p[i].vx*dt;
            p[i].y += p[i].vy*dt;
            p[i].z += p[i].vz*dt;
        }

        hipEventRecord(iter_end, 0);
		hipEventSynchronize(iter_end);

		hipEventElapsedTime(&elapsed_time, iter_start, iter_end);
    	if (iter > 1) { // First iter is warm up
      		totalTime += elapsed_time/1000.0f;
    	}
    	printf("Iteration %d: %.3f seconds\n", iter, elapsed_time/1000.0f);
  	}

  	float avgTime = totalTime / (float)(nIters-1);

	/****************************** Data transfers ******************************/
	hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);

  	printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, 1e-9 * nBodies * nBodies / avgTime);
	printf("Total time: %.3f\n", totalTime);

#ifdef SAVE_FINAL_COORDINATES
	/****************************** Save Final Coordinates ******************************/
	char filename[256];

	sprintf(filename, "cuda_coordinates_%d.txt", nBodies);

	printf("Writing final coordinates to %s\n", filename);
	FILE *fd = fopen(filename, "w");

	if (!fd) {
		perror("Failed opening file");
		return -1;
	}

	for (int i = 0; i < nBodies; i++) {
		fprintf(fd, "%f\n", p[i].x);
		fprintf(fd, "%f\n", p[i].y);
		fprintf(fd, "%f\n", p[i].z);
	}

	fclose(fd);

	printf("Data written successfully\n");
#endif
	
	/****************************** Cleanup ******************************/
	hipFree(d_buf);
	free(buf);
	
	hipDeviceReset();
}