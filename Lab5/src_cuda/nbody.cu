#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"

#define SOFTENING 1e-9f  /* Will guard against denormals */

typedef struct { float x, y, z, vx, vy, vz, Fx, Fy, Fz; } Body;

/****************************** Helper Functions ******************************/
bool checkCudaError(const char *step) {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error in %s: %s\n", step, hipGetErrorString(err));
		return true;
	}
	return false;
}

void randomizeBodies(float *data, int n) {
  	for (int i = 0; i < n; i+=9) {
    	data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
		data[i+1] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
		data[i+2] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
		data[i+3] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
		data[i+4] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
		data[i+5] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  	}
}

/***************** KERNEL CODE *****************/
__global__ void forceComputeKernel(Body *p, int n) {
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	
	float dx, dy, dz;
	float distSqr, invDist, invDist3;
	p[tid].Fx = 0.0f;
	p[tid].Fy = 0.0f;
	p[tid].Fz = 0.0f;

	for (int i = 0; i < n; i++) {
		dx = p[i].x - p[tid].x;
		dy = p[i].y - p[tid].y;
		dz = p[i].z - p[tid].z;
		float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
		float invDist = 1.0f / sqrtf(distSqr);
		float invDist3 = invDist * invDist * invDist;

		p[tid].Fx += dx * invDist3; p[tid].Fy += dy * invDist3; p[tid].Fz += dz * invDist3;
	}
}

__global__ void positionComputeKernel(Body *p, float dt, int n) {
	int i = threadIdx.x + blockIdx.x*blockDim.x;

	p[i].vx += dt*p[i].Fx; p[i].vy += dt*p[i].Fy; p[i].vz += dt*p[i].Fz;

	p[i].x += p[i].vx*dt;
	p[i].y += p[i].vy*dt;
	p[i].z += p[i].vz*dt;
}


int main(const int argc, const char** argv) {

  	int nBodies = 30000;
  	if (argc > 1) nBodies = atoi(argv[1]);

  	const float dt = 0.01f; // time step
  	const int nIters = 10;  // simulation iterations

  	int bytes = nBodies*sizeof(Body);
	float totalTime = 0.0f, elapsed_time = 0.0f;
	float *buf = (float*)malloc(bytes);
	float *d_buf;
	Body *d_p, *p = (Body*)buf;
	hipEvent_t iter_start, iter_end;

	hipEventCreate(&iter_start);
	hipEventCreate(&iter_end);

  	randomizeBodies(buf, 9*nBodies); // Init pos / vel data

	// Set geometry
	dim3 block(1024, 1, 1);
	dim3 grid(ceil(nBodies/1024), 1, 1);

	/****************************** Data transfers ******************************/
	hipMalloc((void **) &d_buf, bytes);
	hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
	d_p = (Body*)d_buf;

	/****************************** Real Computation ******************************/
  	for (int iter = 1; iter <= nIters; iter++) {
		hipEventRecord(iter_start, 0);
		
		forceComputeKernel<<<grid, block>>>(d_p, nBodies);
		hipDeviceSynchronize();
		checkCudaError("forceCoputeKernel");

		positionComputeKernel<<<grid, block>>>(d_p, dt, nBodies);
		
		hipEventRecord(iter_end, 0);
		hipEventSynchronize(iter_end);
		checkCudaError("positionCoputeKernel");

		hipEventElapsedTime(&elapsed_time, iter_start, iter_end);

    	if (iter > 1) { // First iter is warm up
      		totalTime += elapsed_time/1000.0f;
    	}
    	printf("Iteration %d: %.3f seconds\n", iter, elapsed_time/1000.0f);
  	}

  	float avgTime = totalTime / (float)(nIters-1);

	/****************************** Data transfers ******************************/
	hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);
	hipFree(d_buf);

  	printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, 1e-9 * nBodies * nBodies / avgTime);
	printf("Total time: %.3f\n", totalTime);

#ifdef SAVE_FINAL_COORDINATES
	/****************************** Save Final Coordinates ******************************/
	printf("Writing final coordinates to cuda_nbody.txt\n");
	FILE *fd = fopen("cuda_nbody.txt", "w");

	if (!fd) {
		perror("Failed opening file");
		return -1;
	}

	for (int i = 0; i < nBodies; i++) {
		fprintf(fd, "%f\n", p[i].x);
		fprintf(fd, "%f\n", p[i].y);
		fprintf(fd, "%f\n", p[i].z);
	}

	fclose(fd);

	printf("Data written successfully\n");
#endif
	
	free(buf);
	
	hipDeviceReset();
}
