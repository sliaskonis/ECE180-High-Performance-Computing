#include "hip/hip_runtime.h"
extern "C" {
    #include <stdio.h>
    #include <hip/hip_runtime.h>
    #include <time.h>
    #include "hist-equ.h"
    #include "colours.h"

    #define MAX_THREADS_PER_BLOCK 1024
    #define BLOCK_SIZE 256
    #define STRIDE 100
    #define GRID_DIM (ceil((float)img_size/BLOCK_SIZE)/STRIDE)

	/****************************** Helper Functions ******************************/
	bool checkCudaError(const char *step) {
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			printf("Error in %s: %s\n", step, hipGetErrorString(err));
			return true;
		}
		return false;
	}

	/****************************** Kernels ******************************/

    // Histogram calculation: stride implementation
	__global__ void histogram_calc(int *hist_out, unsigned char *img_in, int img_size, int nbr_bin) {
        __shared__ int private_hist[256];

        int i = threadIdx.x + blockIdx.x*blockDim.x,
            stride = blockDim.x * gridDim.x;

        private_hist[threadIdx.x] = 0;
        __syncthreads();
		
        while (i < img_size) {
            atomicAdd(&(private_hist[img_in[i]]), 1);
            i += stride;
        }
        __syncthreads();
        
        atomicAdd(&(hist_out[threadIdx.x]), private_hist[threadIdx.x]);
    }

    // Histogram equalization application: naive implementation
    __global__ void histogram_equ(unsigned char *d_img_in, int *d_lut) {
        int tid = threadIdx.x + blockIdx.x*blockDim.x;
        
        if (d_lut[d_img_in[tid]] > 255) {
            d_img_in[tid] = 255;
        }
        else {
            d_img_in[tid] = (unsigned char)d_lut[d_img_in[tid]];
        }
    }

	// Kernel wrapper
    void histogram_gpu(unsigned char *img_in,
                                int img_size, int nbr_bin) {
        int padding = 0, padded_size = 0;
        float elapsed_time;
        int i, cdf, min, d;
        int *lut = (int *)malloc(sizeof(int)*nbr_bin);
        int *hist_out = (int *)malloc(sizeof(int)*nbr_bin);

		unsigned char *d_img_in;
        int *d_hist_out;
        int *d_lut;

        hipEvent_t gpu_start, gpu_stop, memory_transfers, hist_kernel, hist_equ_kernel_start, hist_equ_kernel_end;
        hipEventCreate(&gpu_start);
        hipEventCreate(&gpu_stop);
        hipEventCreate(&memory_transfers);
        hipEventCreate(&hist_kernel);
        hipEventCreate(&hist_equ_kernel_start);
        hipEventCreate(&hist_equ_kernel_end);


        dim3 block(BLOCK_SIZE, 1, 1);
        dim3 grid(GRID_DIM, 1, 1);

        hipEventRecord(gpu_start, 0);
        /************************* Device Memory Allocation *************************/
        padding = (img_size%MAX_THREADS_PER_BLOCK) ? (MAX_THREADS_PER_BLOCK - (img_size%MAX_THREADS_PER_BLOCK)) : 0;

		padded_size = img_size + padding;
		hipMalloc((void**) &d_img_in,	 sizeof(unsigned char)*padded_size);
        hipMalloc((void**) &d_hist_out, sizeof(int)*nbr_bin);
        hipMalloc((void**) &d_lut,      sizeof(int)*nbr_bin);

        hipMemset (d_img_in,   0, sizeof(unsigned char)*padded_size);
        hipMemset (d_hist_out, 0, sizeof(int)*nbr_bin);
        
		hipMemcpy(d_img_in, img_in, sizeof(unsigned char)*img_size, hipMemcpyHostToDevice);

        hipEventRecord(memory_transfers, 0);

        /************************* Histogram calculation kernel launch *************************/
        histogram_calc<<<grid, block>>>(d_hist_out, d_img_in, img_size, nbr_bin);
		
        hipEventRecord(hist_kernel, 0);
        hipEventSynchronize(hist_kernel);

		checkCudaError("Histogram calculation");

		// Copy calculated histogram back to host 
        hipMemcpy(hist_out, d_hist_out, sizeof(int)*nbr_bin, hipMemcpyDeviceToHost);

        // Clean histogram counts added by the padding elements
		// Padding elements are set to 0
        hist_out[0] = hist_out[0] - padding;
		
        // Construct the LUT by calculating the CDF
        cdf = 0;
        min = 0;
        i = 0;
        while(min == 0){
            min = hist_out[i++];
        }
        d = img_size - min;
        for(i = 0; i < nbr_bin; i ++){
            cdf += hist_out[i];
            lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
            if(lut[i] < 0){
                lut[i] = 0;
            }
        }    
        
        hipMemcpy(d_lut, lut, sizeof(int)*nbr_bin, hipMemcpyHostToDevice);
        
        dim3 block2(MAX_THREADS_PER_BLOCK, 1, 1);
        dim3 grid2(ceil((float)img_size/MAX_THREADS_PER_BLOCK), 1, 1);
        
        hipEventRecord(hist_equ_kernel_start, 0);

        /************************* Histogram equalization kernel launch *************************/
        histogram_equ<<<grid2, block2>>>(d_img_in, d_lut);
        
        hipEventRecord(hist_equ_kernel_end, 0);
        hipEventSynchronize(hist_equ_kernel_end);

        checkCudaError("Histogram equalization");

        // Copy img back to host
        hipMemcpy(img_in, d_img_in, sizeof(unsigned char)*img_size, hipMemcpyDeviceToHost);

        // Free non-wanted memory
        hipFree(d_lut);
        hipFree(d_img_in);
        hipFree(d_hist_out);

        hipEventRecord(gpu_stop, 0);
        hipEventSynchronize(gpu_stop);
        
        // Calculate elapsed time for all events
        hipEventElapsedTime(&elapsed_time, gpu_start, gpu_stop);
        printf( GRN "Total GPU time: %fsec, consists of:\n" RESET, elapsed_time/1000);

        hipEventElapsedTime(&elapsed_time, gpu_start, memory_transfers);
        printf(MAG"\t%f (memory transfers 1)\n" RESET, elapsed_time/1000);

        hipEventElapsedTime(&elapsed_time, memory_transfers, hist_kernel);
        printf(MAG"\t%f (histogram kernel)\n" RESET, elapsed_time/1000);

        hipEventElapsedTime(&elapsed_time, hist_kernel, hist_equ_kernel_start);
        printf(MAG"\t%f (cdf calculation + memory transfers 2)\n" RESET, elapsed_time/1000);

        hipEventElapsedTime(&elapsed_time, hist_equ_kernel_start, hist_equ_kernel_end);
        printf(MAG"\t%f (histogram equalization kernel)\n" RESET, elapsed_time/1000);

        hipEventElapsedTime(&elapsed_time, hist_equ_kernel_end, gpu_stop);
        printf(MAG"\t%f (memory transfers + cleanup)\n" RESET, elapsed_time/1000);

        // Reset the device
        hipDeviceReset();
    }
}