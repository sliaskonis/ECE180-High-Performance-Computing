#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include "hist-equ.h"
#include "colours.h"

void run_cpu_gray_test(PGM_IMG img_in, char *out_filename);
void run_gpu_gray_test(PGM_IMG img_in, char *out_filename);

bool checkCudaError(const char *step) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in %s: %s\n", step, hipGetErrorString(err));
        return true;
    }
    return false;
}

int main(int argc, char *argv[]){
    PGM_IMG img_ibuf_g;

	if (argc != 4) {
		printf("Run with input file name and output file name as arguments\n");
		exit(1);
	}

    img_ibuf_g = read_pgm(argv[1]);

    // printf(YEL "Running contrast enhancement for gray-scale images.\n" RESET);

    // run_cpu_gray_test(img_ibuf_g, argv[2]);

    printf(YEL "\nRunning contrast enhancement for gray-scale images on gpu.\n" RESET);

    run_gpu_gray_test(img_ibuf_g, argv[3]);
    
    free_pgm(img_ibuf_g);

    // Reset the device
    hipDeviceReset();

    return 0;
}

void run_cpu_gray_test(PGM_IMG img_in, char *out_filename)
{
    PGM_IMG img_obuf;

    printf(YEL "Starting CPU processing...\n" RESET);
    img_obuf = contrast_enhancement_cpu(img_in);
    
    write_pgm(img_obuf, out_filename);
    free_pgm(img_obuf);
}

void run_gpu_gray_test(PGM_IMG img_in, char *out_filename)
{
    printf(YEL "Starting GPU processing...\n" RESET);

    histogram_gpu(img_in.img, img_in.w*img_in.h, 256);   
    
    write_pgm(img_in, out_filename);
}

PGM_IMG read_pgm(const char * path){
    FILE * in_file;
    char sbuf[256];

    PGM_IMG result;
    int v_max;//, i;
    in_file = fopen(path, "r");
    if (in_file == NULL){
        printf("Input file not found!\n");
        exit(1);
    }

    if (fscanf(in_file, "%s", sbuf) != 1) {
        fprintf(stderr, "Error reading magic number\n");
        exit(EXIT_FAILURE);
    }
    if (fscanf(in_file, "%d", &result.w) != 1) {
        fprintf(stderr, "Error reading width\n");
        exit(EXIT_FAILURE);
    }
    if (fscanf(in_file, "%d", &result.h) != 1) {
        fprintf(stderr, "Error reading height\n");
        exit(EXIT_FAILURE);
    }
    if (fscanf(in_file, "%d", &v_max) != 1) {
        fprintf(stderr, "Error reading max value\n");
        exit(EXIT_FAILURE);
    }

    printf("Image size: %d x %d\n", result.w, result.h);

    // TODO: try different flags and monitor behaviour
    // hipHostAlloc((void**) &result.img, result.w * result.h * sizeof(unsigned char), hipHostMallocMapped);
    // hipHostAlloc((void**) &result.img, result.w * result.h * sizeof(unsigned char), hipHostMallocWriteCombined); // need mapped too in our case

    hipHostAlloc((void**) &result.img, result.w * result.h * sizeof(unsigned char), hipHostMallocMapped | hipHostMallocWriteCombined);
    
    checkCudaError("hipHostAlloc");

    if (fread(result.img, sizeof(unsigned char), result.w * result.h, in_file) != (size_t)(result.w * result.h)) {
        fprintf(stderr, "Error reading image data\n");
        exit(EXIT_FAILURE);
    }
    fclose(in_file);

    return result;
}

void write_pgm(PGM_IMG img, const char * path){
    FILE * out_file;

    out_file = fopen(path, "wb");
    fprintf(out_file, "P5\n");
    fprintf(out_file, "%d %d\n255\n",img.w, img.h);
    fwrite(img.img,sizeof(unsigned char), img.w*img.h, out_file);
    fclose(out_file);
}

void free_pgm(PGM_IMG img)
{
    hipHostFree(img.img);
    checkCudaError("hipHostFree");
}

