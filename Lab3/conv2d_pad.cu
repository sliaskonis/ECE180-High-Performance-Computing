/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy    0.05

/***************************************
 *   Reference Row Convolution Filter  *
 ***************************************/
__global__ void convolutionRowGPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {
    int tx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int ty = (blockIdx.y * blockDim.y) + threadIdx.y;
    float sum=0;

      for (int k = -filterR; k <= filterR; k++) {
        int d = tx + k;
        sum += h_Src[ty * imageW + d] * h_Filter[filterR - k];     
      }
      h_Dst[ty * imageW + tx] = sum;  
  }

/******************************************
 *   Reference Column Convolution Filter  *
 ******************************************/
__global__ void convolutionColumnGPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {
    int tx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int ty = (blockIdx.y * blockDim.y) + threadIdx.y;
    float sum=0;

      for (int k = -filterR; k <= filterR; k++) {
        int d = ty + k;
        sum += h_Src[d * imageW + tx] * h_Filter[filterR - k];     
      }
      h_Dst[ty * imageW + tx] = sum;  
}

// Reference row convolution filter
__host__ void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = filterR; y < imageH + filterR; y++) {
    for (x = filterR; x < imageW + filterR; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;
        sum += h_Src[y * imageW + d] * h_Filter[filterR - k];   
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
}

// Reference column convolution filter
__host__ void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = filterR; y < imageH + filterR; y++) {
    for (x = filterR; x < imageW + filterR; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;
        sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
}

// Main program
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;

    float
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU;

    int imageW;
    int imageH;
    unsigned int i;

    // We assume that imageW = imageH = N, where N is given by the user.
    if (argc != 3) {
        printf("Usage: %s <image size> <filter radius>\n", argv[0]);
        printf("Image size must be a power of 2\n");
        exit(1);
    }

    imageW = atoi(argv[1]);
    filter_radius = atoi(argv[2]);
    imageH = imageW;

    if (imageW < FILTER_LENGTH) {
        printf("Error: Filter length exceeds image dimensions\n");
        exit(1);
    }

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");

    int padded_size = (imageW + 2*filter_radius) * (imageH + 2*filter_radius);

    /************************ Host memory allocation ************************/
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    assert(h_Filter != NULL);
    h_Input     = (float *)calloc(padded_size , sizeof(float));
    assert(h_Input != NULL);
    h_Buffer    = (float *)calloc(padded_size , sizeof(float));
    assert(h_Buffer != NULL);
    h_OutputCPU = (float *)calloc(padded_size , sizeof(float));
    assert(h_OutputCPU != NULL);
    h_OutputGPU = (float *)calloc(padded_size , sizeof(float));
    assert(h_OutputGPU != NULL);

    /************************ Device memory allocation ************************/
    hipMalloc((void**) &d_Filter, FILTER_LENGTH*sizeof(float));
    hipMalloc((void**) &d_Input, padded_size*sizeof(float));
    hipMalloc((void**) &d_Buffer, padded_size*sizeof(float));
    hipMalloc((void**) &d_OutputGPU, padded_size*sizeof(float));

    // Initialize Filter and Image.
    // Both filter and image are stored in row-major order and are initialized
    // with random values.
    srand(200);
    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = filter_radius; i < imageH + filter_radius; i++) {
        for (int j = filter_radius; j < imageW + filter_radius; j++) {
            h_Input[i*(imageW+(2*filter_radius))+j] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
        }
    }

    /**********************  Copy Memory to Device ***************************/
    hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_Input, h_Input, padded_size*sizeof(float),hipMemcpyHostToDevice);
    
    /********************************** CPU Execution **********************************/
    printf("CPU computation...\n");

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius);          // Row convolution       
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius);   // Column convolution

    /********************************** GPU Execution **********************************/
    printf("GPU computation...\n");

    /**********************  Kernel Launch Configuration ***************************/
    int gridDimx = ceil((float)imageW/32);
    int gridDimy = ceil((float)imageH/32);
    dim3 dimGrid(gridDimx, gridDimy);
    dim3 dimBlock(32, 32);

    // if (gridDimx == 1) { dim3 dimBlock(imageW, imageH); }
    // else { dim3 dimBlock(32, 32); }

    printf("Grid: %d x %d\n", gridDimx, gridDimy);
    printf("Block: %d x %d\n", imageW, imageH);

    convolutionRowGPU<<<dimGrid, dimBlock>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in convolutionRowGPU: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    convolutionColumnGPU<<<dimGrid, dimBlock>>>(d_OutputGPU, d_Buffer, d_Filter, imageW, imageH, filter_radius);
    
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in convolutionColumnGPU: %s\n", hipGetErrorString(err));
    }
    else {
        // Copy results from device to host
        hipMemcpy(h_OutputGPU, d_OutputGPU, padded_size*sizeof(float),hipMemcpyDeviceToHost);

        /********************** Verify Correctness **********************/
        printf("Verifying results...\n");

        int errors = 0;
        for (i = filter_radius; i < imageH + filter_radius; i++) {
            for (int j = filter_radius; j < imageW + filter_radius; j++) {
                float error = ABS(h_OutputCPU[i*imageW+j] - h_OutputGPU[i*imageW+j]);
                if (error > accuracy) {
                    errors++;
                    printf("Mismatch at index %d: CPU = %f, GPU = %f, Error = %f\n", 
                        i*imageW+j, h_OutputCPU[i*imageW+j], h_OutputGPU[i*imageW+j], error);
                }
            }
        }

        if (errors == 0) {
            printf("TEST PASSED\n");
        } else {
            printf("TEST FAILED with %d errors\n", errors);
        }
    }

    // Free Host allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    
    // Free Device allocated memory
    hipFree(d_Filter);
    hipFree(d_Input);
    hipFree(d_OutputGPU);

    // Reset the device and exit
    hipDeviceReset();

    return 0;
}