#include "hip/hip_runtime.h"
/******************************************************************************
 *                                                                            *
 *  File:       conv2d_pad.cu                                                *
 *  Description:                                                              *
 *      This file contains the implementation of a 2D convolution using       *  
 *      a separable filter. The convolution is performed on the CPU as well   *
 *      as the GPU using CUDA.                                                *
 *      For the GPU implementation, the geometry used is a 2D grid of 2D      *
 *      thread blocks.                                                        *
 *                                                                            *
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)

/****************** ACCURACY MACROS ******************/
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy    0.5

/****************** GRID/BLOCK GEOMETRY ******************/
#define TILE_WIDTH 32
#define GRID_X ceil((float)imageW/TILE_WIDTH)
#define GRID_Y ceil((float)imageH/TILE_WIDTH)

bool checkCudaError(const char *step) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in %s: %s\n", step, hipGetErrorString(err));
        return true;
    }
    return false;
}

/*
------------------------------ CPU/GPU CONVOLUTION FUNCTION DECLARATION ------------------------------
*/

/***************************************
 *           GPU Row Convolution       *
 ***************************************/
__global__ void convolutionRowGPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {
    int tx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int ty = (blockIdx.y * blockDim.y) + threadIdx.y;
    float sum=0;

    int xnew = tx + filterR;
    int ynew = ty + filterR;

    for (int k = -filterR; k <= filterR; k++) {
        int d = xnew + k;
        sum += h_Src[ynew * imageW + d] * h_Filter[filterR - k];
    }

    h_Dst[ynew * imageW + xnew] = sum;
}

/***************************************
 *           GPU Column Convolution    *
 ***************************************/
__global__ void convolutionColumnGPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {
    int tx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int ty = (blockIdx.y * blockDim.y) + threadIdx.y;
    float sum=0;

    int xnew = tx + filterR;
    int ynew = ty + filterR;

    for (int k = -filterR; k <= filterR; k++) {
        int d = ynew + k;
        sum += h_Src[d * imageW + xnew] * h_Filter[filterR - k];
    }

    h_Dst[ynew * imageW + xnew] = sum;
}

/***************************************
 *           CPU Row Convolution       *
 ***************************************/
__host__ void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = filterR; y < imageH - filterR; y++) {
    for (x = filterR; x < imageW - filterR; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;
        sum += h_Src[y * imageW + d] * h_Filter[filterR - k];   
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
}

/***************************************
 *           CPU Row Convolution       *
 ***************************************/
__host__ void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = filterR; y < imageH - filterR; y++) {
    for (x = filterR; x < imageW - filterR; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;
        sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
}

/*
------------------------------ END OF FUNCTION DECLARATION ------------------------------
*/

int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;

    float
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU;

    int imageW;
    int imageH;
    unsigned int i;

    bool err;

    // We assume that imageW = imageH = N, where N is given by the user.
    if (argc != 3) {
        printf("Usage: %s <image size> <filter radius>\n", argv[0]);
        printf("Image size must be a power of 2\n");
        exit(1);
    }

    imageW = atoi(argv[1]);
    filter_radius = atoi(argv[2]);
    imageH = imageW;

    if (imageW < FILTER_LENGTH) {
        printf("Error: Filter length exceeds image dimensions\n");
        exit(1);
    }

    int padded_size = (imageW + 2*filter_radius) * (imageH + 2*filter_radius);
    int newImageW = imageW + (2 * filter_radius);
    int newImageH = imageH + (2 * filter_radius);

    /**********************************************************/
    /*                   Host Memory Allocation               */
    /**********************************************************/
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    assert(h_Filter != NULL);
    h_Input     = (float *)calloc(padded_size , sizeof(float));
    assert(h_Input != NULL);
    h_Buffer    = (float *)calloc(padded_size , sizeof(float));
    assert(h_Buffer != NULL);
    h_OutputCPU = (float *)calloc(padded_size , sizeof(float));
    assert(h_OutputCPU != NULL);
    h_OutputGPU = (float *)calloc(padded_size , sizeof(float));
    assert(h_OutputGPU != NULL);

    /**********************************************************/
    /*                   Memory Initialization                */
    /**********************************************************/
    srand(200);
    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (int i = 0; i < imageH; i++) {
        for (int j = 0; j < imageW; j++) {
            h_Input[(i + filter_radius) * (imageW + 2 * filter_radius) + (j + filter_radius)] = 
                (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
        }
    }

    /**********************************************************/
    /*                   Host Code Execution                  */
    /**********************************************************/

    clock_t start = clock();
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, newImageW, newImageH, filter_radius);          // Row convolution       
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, newImageW, newImageH, filter_radius);   // Column convolution
    clock_t end = clock();

    double cpu_time = ((double) (end - start)) / CLOCKS_PER_SEC * 1000;

    /**********************************************************/
    /*                   Device Memory Allocation             */
    /**********************************************************/
    hipMalloc((void**) &d_Filter, FILTER_LENGTH*sizeof(float));
    hipMalloc((void**) &d_Input, padded_size*sizeof(float));
    hipMalloc((void**) &d_Buffer, padded_size*sizeof(float));
    hipMalloc((void**) &d_OutputGPU, padded_size*sizeof(float));

    hipMemset(d_Buffer, 0, padded_size*sizeof(float));
    hipMemset(d_OutputGPU, 0, padded_size*sizeof(float));

    /**********************************************************/
    /*                   Device Code Execution                */
    /**********************************************************/

    dim3 dimGrid(GRID_X, GRID_Y);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    hipEvent_t startGPU, stopGPU;
    float gpu_time;

    // Create CUDA events for timing purposes
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);

    // Start GPU timing
    hipEventRecord(startGPU);
    
    // Copy filter and input data to device
    hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_Input, h_Input, padded_size*sizeof(float),hipMemcpyHostToDevice);

    // Perform row-wise convolution on the GPU
    convolutionRowGPU<<<dimGrid, dimBlock>>>(d_Buffer, d_Input, d_Filter, newImageW, newImageH, filter_radius);  
    err = checkCudaError("convolutionRowGPU");
    
    // Synchronize the device to ensure the row convolution is complete
    hipDeviceSynchronize();
    
    // Perform column-wise convolution on the GPU
    convolutionColumnGPU<<<dimGrid, dimBlock>>>(d_OutputGPU, d_Buffer, d_Filter, newImageW, newImageH, filter_radius);
    err = checkCudaError("convolutionColumnGPU");

    // Copy the resulting data back to the host
    hipMemcpy(h_OutputGPU, d_OutputGPU, padded_size*sizeof(float),hipMemcpyDeviceToHost);

    // Stop GPU timing
    hipEventRecord(stopGPU);
    hipEventSynchronize(stopGPU);

    // Calculate GPU execution time
    hipEventElapsedTime(&gpu_time, startGPU, stopGPU);
    
    // Destroy CUDA events
    hipEventDestroy(startGPU);
    hipEventDestroy(stopGPU);

    /**********************************************************/
    /*                    Verify Correctness                  */
    /**********************************************************/
#ifdef VERIFY    
    if (!err) {
        
        printf("Verifying results...\n");

        int errors = 0;
        for (i = filter_radius; i < newImageH-filter_radius; i++) {
            for (int j = filter_radius; j < newImageW - filter_radius; j++) {
                float error = ABS(h_OutputCPU[i*newImageH+j] - h_OutputGPU[i*newImageH+j]);
                if (error > accuracy) {
                    errors++;
                    printf("Mismatch at index %d: CPU = %f, GPU = %f, Error = %f\n", 
                        i, h_OutputCPU[i], h_OutputGPU[i], error);
                }
            }
        }

        if (errors == 0) {
            printf("TEST PASSED\n");
        } else {
            printf("TEST FAILED with %d errors\n", errors);
        }
    }
#endif

#ifdef PRINT_TIMING
    printf("CPU Execution Time: %f\n", cpu_time);
    printf("GPU Execution Time: %f\n", gpu_time);
#endif

    // Free Host allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    
    // Free Device allocated memory
    hipFree(d_Filter);
    hipFree(d_Input);
    hipFree(d_OutputGPU);

    // Reset the device and exit
    hipDeviceReset();

    return 0;
}